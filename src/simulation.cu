#include "hip/hip_runtime.h"
#include <cstdio>

#include "glad/glad.h"
#include <cuda_gl_interop.h>

#include "global_defines.h"
#include "simulation.hpp"

#define GRID_SIZE SIM_GRID_SIZE
#define GRID_ELEMENTS (GRID_SIZE*GRID_SIZE*GRID_SIZE)
#define BLOCK_SIZE 8

#define DENSITY_TEXTURE_SIZE (GRID_ELEMENTS*sizeof(float))

// https://stackoverflow.com/a/14038590/11617929
#define cudaCheckErrors(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true) {
   if (code != hipSuccess) {
      fprintf(stderr,"CUDA Assert %s(%d): %s\n", file, line, hipGetErrorString(code));
      if (abort) exit(code);
   }
}

long sim_frame_counter = 0;
// tick on even frames: read from tick buffer, write to tock buffer
__host__
inline bool sim_frame_is_tick() {
    return sim_frame_counter % 2 == 0;
}
// tock on odd frames:  read from tock buffer, write to tick buffer
__host__
inline bool sim_frame_is_tock() {
    return sim_frame_counter % 2 != 0;
}

void set_volume_texture_parameters(textureReference* texture) {
    texture->normalized = true;
    texture->addressMode[0] = texture->addressMode[1] = texture->addressMode[2] = hipAddressModeWrap;
    texture->minMipmapLevelClamp = texture->maxMipmapLevelClamp = 0.0;
    texture->mipmapFilterMode = hipFilterModePoint;
}

// CUDA + GL 3D TEXTURE INTEROP STRATEGY
//
// Start of program:
// 1. Init texture the normal OpenGL way
// 2. Register texture with CUDA as a graphics resource
//  - Need to specify surface read/write flag
//
// Every frame:
// 3. Map the texture resource to CUDA
//  - Must not be accessed by OpenGL while mapped
// 4. Get a CUDA array pointer from the resource
// 5. Bind array to surface reference
// 6. Write to surface in kernel
// 7. Unmap the texture resource
//  - OpenGL can access the texture again
//
// End of program:
// 8. Unregister the texture

// DENSITY FIELD
// OpenGL interop
GLuint                 gl_density_texture;
hipGraphicsResource_t gl_density_texture_resource;

// kernel read/write targets
surface<void,  cudaSurfaceType3D>                          d_density_write_surface;
texture<float, hipTextureType3D, hipReadModeElementType> d_density_read_texture;

// array to map gl_density_texture onto
hipArray_t d_density_texture_mapped_array;
// backing array for d_density_read_texture
hipArray_t d_density_read_array; // TODO: experiment with copy-free (double-buffer) implementation

// VELOCITY FIELD
// backing velocity field double buffers
hipArray_t d_velocity_tick_array;
hipArray_t d_velocity_tock_array;

// kernel read/write targets
// backing arrays are swapped between tick and tock
surface<void,   cudaSurfaceType3D>                          d_velocity_write_surface;
texture<float4, hipTextureType3D, hipReadModeElementType> d_velocity_read_texture;
// TODO: find a use for the 4th velocity component

// DEBUG DATA FIELD
// TODO: elide in release builds
SimDebugDataMode sim_debug_data_mode            = None;
SimDebugDataMode sim_debug_data_mode_prev_frame = None;
__constant__
SimDebugDataMode d_debug_data_mode;

// OpenGL interop
GLuint                 gl_debug_data_texture;
hipGraphicsResource_t gl_debug_data_texture_resource;

surface<void, cudaSurfaceType3D> d_debug_data_write_surface;

// INITIALIZATION

__host__
void sim_init(GLenum density_texture_unit, GLenum debug_data_texture_unit) {
    // OPENGL-HOSTED DENSITY TEXTURE
    glActiveTexture(density_texture_unit);
    glGenTextures(1, &gl_density_texture);
    glBindTexture(GL_TEXTURE_3D, gl_density_texture);

    // texture parameters
    glTexParameteri(GL_TEXTURE_3D, GL_TEXTURE_WRAP_R, GL_CLAMP_TO_BORDER);
    glTexParameteri(GL_TEXTURE_3D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_BORDER);
    glTexParameteri(GL_TEXTURE_3D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_BORDER);

    glTexParameteri(GL_TEXTURE_3D, GL_TEXTURE_MAX_LEVEL, 0);

    glTexParameteri(GL_TEXTURE_3D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
    glTexParameteri(GL_TEXTURE_3D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);

    // allocate texture storage
    glTexImage3D(
        GL_TEXTURE_3D, 0, GL_R32F,
        GRID_SIZE, GRID_SIZE, GRID_SIZE, 0,
        GL_RED, GL_FLOAT, NULL
    );

    // get a handle to the texture for CUDA
    cudaCheckErrors(hipGraphicsGLRegisterImage(
        &gl_density_texture_resource, gl_density_texture,
        GL_TEXTURE_3D,
        // flag to enable surface writing to arrays
        hipGraphicsRegisterFlagsSurfaceLoadStore
    ));
    glActiveTexture(0);

    // initialize CUDA's helper buffer d_density_read_array
    hipChannelFormatDesc density_format = hipCreateChannelDesc<float>();
    hipExtent density_extent = make_hipExtent(GRID_SIZE, GRID_SIZE, GRID_SIZE);
    cudaCheckErrors(hipMalloc3DArray(
        &d_density_read_array, &density_format, density_extent,
        hipArraySurfaceLoadStore
    ));
    cudaCheckErrors(hipBindTextureToArray(
        d_density_read_texture, d_density_read_array
    ));
    set_volume_texture_parameters(&d_density_read_texture);


    // CUDA-HOSTED VELOCITY FIELD BUFFERS
    hipChannelFormatDesc velocity_format = hipCreateChannelDesc<float4>();
    hipExtent velocity_extent = make_hipExtent(GRID_SIZE, GRID_SIZE, GRID_SIZE);
    cudaCheckErrors(hipMalloc3DArray(
        &d_velocity_tick_array, &velocity_format, velocity_extent,
        hipArraySurfaceLoadStore
    ));
    cudaCheckErrors(hipMalloc3DArray(
        &d_velocity_tock_array, &velocity_format, velocity_extent,
        hipArraySurfaceLoadStore
    ));
    set_volume_texture_parameters(&d_velocity_read_texture);


    // OPENGL-HOSTED DEBUG TEXTURE
    glActiveTexture(debug_data_texture_unit);
    glGenTextures(1, &gl_debug_data_texture);
    glBindTexture(GL_TEXTURE_3D, gl_debug_data_texture);

    // texture parameters
    glTexParameteri(GL_TEXTURE_3D, GL_TEXTURE_WRAP_R, GL_CLAMP_TO_BORDER);
    glTexParameteri(GL_TEXTURE_3D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_BORDER);
    glTexParameteri(GL_TEXTURE_3D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_BORDER);

    glTexParameteri(GL_TEXTURE_3D, GL_TEXTURE_MAX_LEVEL, 0);

    glTexParameteri(GL_TEXTURE_3D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_3D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);

    // allocate texture storage
    glTexImage3D(
        GL_TEXTURE_3D, 0, GL_RGBA32F,
        GRID_SIZE, GRID_SIZE, GRID_SIZE, 0,
        GL_RGBA, GL_FLOAT, NULL
    );

    // get a handle to the texture for CUDA
    cudaCheckErrors(hipGraphicsGLRegisterImage(
        &gl_debug_data_texture_resource, gl_debug_data_texture,
        GL_TEXTURE_3D,
        // flag to enable surface writing to arrays
        hipGraphicsRegisterFlagsSurfaceLoadStore
    ));
    glActiveTexture(0);
}

__host__
void sim_terminate() {
    cudaCheckErrors(hipGraphicsUnregisterResource(gl_density_texture_resource));
    glDeleteTextures(1, &gl_density_texture);
}

// RESOURCE MAP & UNMAP PROCEDURES

__host__
void sim_map_gl_density() {
    // borrow resource from GL and bind to surface reference
    cudaCheckErrors(hipGraphicsMapResources(1, &gl_density_texture_resource, 0));
    cudaCheckErrors(hipGraphicsSubResourceGetMappedArray(
        &d_density_texture_mapped_array, gl_density_texture_resource,
        0, 0
    ));
    // bind the texture's array to the surface reference, enabling writing
    cudaCheckErrors(cudaBindSurfaceToArray(
        d_density_write_surface, d_density_texture_mapped_array
    ));
}

__host__
void sim_unmap_gl_density_without_updating_read_array() {
    cudaCheckErrors(hipGraphicsUnmapResources(1, &gl_density_texture_resource, 0));
}

__host__
void sim_unmap_gl_density() {
    // TODO: experiment with copy direction and async
    hipMemcpy3DParms copy = {0};
    copy.srcArray = d_density_texture_mapped_array;
    copy.dstArray = d_density_read_array;
    copy.extent   = make_hipExtent(GRID_SIZE, GRID_SIZE, GRID_SIZE);
    copy.kind     = hipMemcpyDeviceToDevice;
    cudaCheckErrors(hipMemcpy3D(&copy));
    sim_unmap_gl_density_without_updating_read_array();
}

__host__
void sim_map_gl_debug_data() {
    // borrow resource from GL and bind to surface reference
    cudaCheckErrors(hipGraphicsMapResources(1, &gl_debug_data_texture_resource, 0));
    hipArray_t d_debug_data_texture_mapped_array;
    cudaCheckErrors(hipGraphicsSubResourceGetMappedArray(
        &d_debug_data_texture_mapped_array, gl_debug_data_texture_resource,
        0, 0
    ));
    // bind the texture's array to the surface reference, enabling writing
    cudaCheckErrors(cudaBindSurfaceToArray(
        d_debug_data_write_surface, d_debug_data_texture_mapped_array
    ));
}

__host__
void sim_unmap_gl_debug_data() {
    cudaCheckErrors(hipGraphicsUnmapResources(1, &gl_debug_data_texture_resource, 0));
}

// API FUNCTIONS & KERNELS

__global__
void sim_update_kernel(double dt) {
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    int z = blockIdx.z*blockDim.z + threadIdx.z;

    float px = ((float) x / (float) (GRID_SIZE-1));
    float py = ((float) y / (float) (GRID_SIZE-1));
    float pz = ((float) z / (float) (GRID_SIZE-1));
    float4 v = tex3D(
        d_velocity_read_texture,
        px, py, pz
    );
    // TODO: find q using a real path integrator
    float qx = px - v.x * dt / (float) (GRID_SIZE-1);
    float qy = py - v.y * dt / (float) (GRID_SIZE-1);
    float qz = pz - v.z * dt / (float) (GRID_SIZE-1);

    // advection
    float4 w = tex3D(
        d_velocity_read_texture,
        qx, qy, qz
    );
    surf3Dwrite(
        w, d_velocity_write_surface,
        x*sizeof(float4), y, z,
        hipBoundaryModeTrap
    );

    // substance transport
    float density = tex3D(
        d_density_read_texture,
        qx, qy, qz
    );
    surf3Dwrite(
        density, d_density_write_surface,
        x*sizeof(float), y, z,
        hipBoundaryModeTrap
    );

    // TODO: eliminate conditional by moving debug_data_mode/bool into template?
    if (d_debug_data_mode) {
        float4 debug_data;
        switch (d_debug_data_mode) {
        case NormalizedVelocityAndMagnitude: {
            float l = sqrt(v.x*v.x + v.y*v.y + v.z*v.z);
            debug_data = make_float4(v.x/l, v.y/l, v.z/l, l);
            break;
        }
        default:
            debug_data = make_float4(NAN, NAN, NAN, NAN);
            break;
        }
        surf3Dwrite(
            debug_data, d_debug_data_write_surface,
            x*sizeof(float4), y, z,
            hipBoundaryModeTrap
        );
    }
}

// TODO: constant dt
__host__
void sim_update(double dt) {
    sim_map_gl_density();

    if (sim_debug_data_mode != sim_debug_data_mode_prev_frame) {
        hipMemcpyToSymbol(HIP_SYMBOL(d_debug_data_mode), &sim_debug_data_mode, sizeof(SimDebugDataMode));
        sim_debug_data_mode_prev_frame = sim_debug_data_mode;
    }
    if (sim_debug_data_mode) sim_map_gl_debug_data();
    // TODO: function to write debug_data without stepping simulating (to switch views while sim paused)

    // set read and write buffers
    if (sim_frame_is_tick()) {
        cudaCheckErrors(hipBindTextureToArray(d_velocity_read_texture,  d_velocity_tick_array));
        cudaCheckErrors(cudaBindSurfaceToArray(d_velocity_write_surface, d_velocity_tock_array));
    } else {
        cudaCheckErrors(hipBindTextureToArray(d_velocity_read_texture,  d_velocity_tock_array));
        cudaCheckErrors(cudaBindSurfaceToArray(d_velocity_write_surface, d_velocity_tick_array));
    }

    dim3 blocks  = dim3(GRID_SIZE/BLOCK_SIZE, GRID_SIZE/BLOCK_SIZE, GRID_SIZE/BLOCK_SIZE);
    dim3 threads = dim3(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE);
    sim_update_kernel<<<blocks, threads>>>(dt);

    if (sim_debug_data_mode) sim_unmap_gl_debug_data();
    sim_unmap_gl_density();
    sim_frame_counter += 1;
}

// DEBUG FUNCTIONS

__global__
void sim_debug_reset_velocity_field_kernel(float3 time) {
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    int z = blockIdx.z*blockDim.z + threadIdx.z;

    float px = ((float) x / (float) (GRID_SIZE-1))*2.0 - 1.0;
    float py = ((float) y / (float) (GRID_SIZE-1))*2.0 - 1.0;
    float pz = ((float) z / (float) (GRID_SIZE-1))*2.0 - 1.0;

    const float fade_radius = 0.8f;

    float ts[3] = {time.x, time.y, time.z};
    float v[3];
    for (int i = 0; i < 3; i++) {
        // sphere fade
        float d = fmaxf(0.0, (sqrt(px*px + py*py + pz*pz) - fade_radius) / (1-fade_radius));
        float fade = fmin(1.0, 1.0 - sqrt(d));

        float t = 0.3f*ts[i];
        px -= 0.25;
        pz -= 0.25;
        float l = sqrt(px*px + pz*pz);
        float wave = sinf(0.15*TAU*(py + 1.5*sinf(TAU*(0.5*l + 0.3*t)) + 0.15*(1.0+0.2*sinf(TAU*0.4*t))*0.2*t));

        v[i] = (float) (15.0 * wave);
    }
    float4 velocity = make_float4(v[0], v[1], v[2], 0.0);
    surf3Dwrite(
        velocity, d_velocity_write_surface,
        x*sizeof(float4), y, z,
        hipBoundaryModeTrap
    );
}

__host__
void sim_debug_reset_velocity_field(double tx, double ty, double tz) {
    // TODO: decide correct buffer to write into
    if (sim_frame_is_tick()) {
        cudaCheckErrors(cudaBindSurfaceToArray(d_velocity_write_surface, d_velocity_tick_array));
    } else {
        cudaCheckErrors(cudaBindSurfaceToArray(d_velocity_write_surface, d_velocity_tock_array));
    }

    dim3 blocks  = dim3(GRID_SIZE/BLOCK_SIZE, GRID_SIZE/BLOCK_SIZE, GRID_SIZE/BLOCK_SIZE);
    dim3 threads = dim3(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE);
    sim_debug_reset_velocity_field_kernel<<<blocks, threads>>>(make_float3(tx, ty, tz));
}

__global__
void sim_debug_reset_density_field_kernel(double time) {
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    int z = blockIdx.z*blockDim.z + threadIdx.z;

    float px = ((float) x / (float) (GRID_SIZE-1))*2.0 - 1.0;
    float py = ((float) y / (float) (GRID_SIZE-1))*2.0 - 1.0;
    float pz = ((float) z / (float) (GRID_SIZE-1))*2.0 - 1.0;

    const float fade_radius = 0.8f;

    // sphere fade
    float d = fmaxf(0.0, (sqrt(px*px + py*py + pz*pz) - fade_radius) / (1-fade_radius));
    float fade = fmin(1.0, 1.0 - sqrt(d));

    float t = 0.3*time;
    px -= 0.25;
    pz -= 0.25;
    float l = sqrt(px*px + pz*pz);
    float wave = sinf(TAU*(py + 1.5*sinf(TAU*(0.5*l + 0.3*t)) + 0.15*(1.0+0.2*sinf(TAU*0.4*t))*0.2*t));

    float density = 0.7 * fade * (fmaxf(0.0, wave));
    surf3Dwrite(
        density, d_density_write_surface,
        x*sizeof(float), y, z,
        hipBoundaryModeTrap
    );
}

__host__
void sim_debug_reset_density_field(double t) {
    sim_map_gl_density();

    dim3 blocks  = dim3(GRID_SIZE/BLOCK_SIZE, GRID_SIZE/BLOCK_SIZE, GRID_SIZE/BLOCK_SIZE);
    dim3 threads = dim3(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE);
    sim_debug_reset_density_field_kernel<<<blocks, threads>>>(t);

    sim_unmap_gl_density();
}
